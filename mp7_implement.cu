#include "hip/hip_runtime.h"
// Histogram Equalization

#include    <wb.h>

#define HISTOGRAM_LENGTH 256
#define BLOCK_WIDTH 32
#define CHANNEL 3

int ceil(int a, int b){
    return (a + b - 1) / b;
}
//@@ insert code here

// pscan
__global__ void cal_cdf(unsigned int * inputHist, unsigned int * cdf) {

    /*
        calculate cdf 
    */
    __shared__ unsigned int shared_data[HISTOGRAM_LENGTH];

    int tid = threadIdx.x;    

    // each thread load 1 element
    if(tid < HISTOGRAM_LENGTH){
        shared_data[tid] = inputHist[tid];
    }
    __syncthreads();

    // up-sweep phase 

    int offset = 1;
    for(int d = HISTOGRAM_LENGTH / 2; d > 0; d /= 2){
        __syncthreads();
        if(tid < d){
            int bi = offset * 2 * (tid + 1) - 1;
            int ai = bi - offset;
            shared_data[bi] += shared_data[ai];
        }
        offset *= 2;
       
    }
    __syncthreads();

    // clear last element to zero and save it to block_sum
    if(tid == 0){
        shared_data[HISTOGRAM_LENGTH - 1] = 0;
        
    }

    __syncthreads();

    // down-sweep phase
    for(int d = 1; d < HISTOGRAM_LENGTH; d *= 2){
        offset >>= 1;
        __syncthreads();
        if(tid < d){
            int bi = offset * 2 * (tid + 1) - 1;
            int ai = bi - offset;
            float t = shared_data[ai];
            shared_data[ai] = shared_data[bi];
            shared_data[bi] += t;
        } 
        
    }
    __syncthreads();
    
    // here we get exclusive prefix sum, we add them with original data to get inclusive prefix sum
    if(tid < HISTOGRAM_LENGTH){
        cdf[tid] = inputHist[tid] + shared_data[tid];
    }
}

__global__ void histogram_equalization(float * deviceInputImage, float* deviceOutputImage, unsigned int* cdf, int width, int height){
    
    //
    int by = blockIdx.y;
    int bx = blockIdx.x;
    //
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // coordinate
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    int channel = blockIdx.z;

    if(row < height && col < width){
        int val = (unsigned char)(255 * deviceInputImage[(row * width + col) * CHANNEL + channel]);
        deviceOutputImage[(row * width + col) * CHANNEL + channel] = ((unsigned char)(255.0*(cdf[val] - cdf[0])/(cdf[HISTOGRAM_LENGTH - 1] - cdf[0]))) / 255.0;
    }
}


// calculate hist
__global__ void hist(unsigned char* inputImage, int length, unsigned int* hist_output){
    __shared__ unsigned int hist[HISTOGRAM_LENGTH];
    // init 
    if(threadIdx.x < HISTOGRAM_LENGTH){
        hist[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // accumulate 
    int pixel = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while(pixel < length){
        atomicAdd(&(hist[inputImage[pixel]]), 1);
        pixel += stride;
    }
    __syncthreads();
    // copy output to global memory
    if(threadIdx.x < 256){
        atomicAdd(&(hist_output[threadIdx.x]), hist[threadIdx.x]);
    }
}

//
__global__ void cast_and_convert(float* inputImage, unsigned char* outputImage, int height, unsigned int width){
    // get block corrdination 
    int bx = blockIdx.x;
    int by = blockIdx.y;
    // get thead coordination
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // pixel = blockId * BlockSize + threadId
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    // cast data type
    if(row < height && col < width){
        float res = 0.0;
        res += 0.21 * (unsigned char)(255 * inputImage[(row * width + col) * CHANNEL]);
        res += 0.71 * (unsigned char)(255 * inputImage[(row * width + col) * CHANNEL + 1]);
        res += 0.07 * (unsigned char)(255 * inputImage[(row * width + col) * CHANNEL + 2]);
        outputImage[row * width + col] = (unsigned char)res;
        
    }
    

}

int main(int argc, char ** argv) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    int imageChannels;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    const char * inputImageFile;

    //@@ Insert more code here

    float * deviceInputImageData;
    unsigned char * deviceInputImageDataGray;
    float * deviceOutputImageData; 
    unsigned int *  deviceHist;
    unsigned int *  deviceCDF;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbPPM_import(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    // -3. initialize hostInputImageData and hostOutputImageData
    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    //@@ insert code here
    
    // -2. allocate memmory on GPU
    hipMalloc((void **)&deviceInputImageData, sizeof(float) * imageHeight * imageWidth * imageChannels);
    hipMalloc((void **)&deviceInputImageDataGray, sizeof(unsigned char) * imageHeight * imageWidth);
    hipMalloc((void **)&deviceOutputImageData, sizeof(float) * imageHeight * imageWidth * imageChannels);
    hipMalloc((void **)&deviceHist, sizeof(unsigned int) * HISTOGRAM_LENGTH);
    hipMalloc((void **)&deviceCDF, sizeof(unsigned int) * HISTOGRAM_LENGTH);

    // -1. copy memory to GPU
    hipMemcpy(deviceInputImageData, hostInputImageData, sizeof(float) * imageHeight * imageWidth * imageChannels, hipMemcpyHostToDevice);

    // 0. do GPU computation
    dim3 DimGrid1(ceil(imageWidth, BLOCK_WIDTH), ceil(imageHeight, BLOCK_WIDTH), 1);
    dim3 DimBlock1(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    std::cout<<"check input "<<std::endl;
    for(int row = 0; row < 5; row ++){
        for(int col = 0; col < 5; col ++){
            std::cout<<hostInputImageData[(row * imageWidth + col) * imageChannels + 0]<<", ";
        }
        std::cout<<endl;
    }

    // 1. cast float to unsigned char
    cast_and_convert<<<DimGrid1, DimBlock1>>>(deviceInputImageData, deviceInputImageDataGray, imageHeight, imageWidth);
    // TODO: This is for debugging
    /*
    hipDeviceSynchronize();
    unsigned char* hostInputImageDataGray = (unsigned char*) malloc(imageHeight * imageWidth * sizeof(unsigned char*));
    hipMemcpy(hostInputImageDataGray, deviceInputImageDataGray, imageHeight * imageWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);
    std::cout<<"check gray image "<<std::endl;
    for(int row = 0; row < 5; row ++){
        for(int col = 0; col < 5; col ++){
            std::cout<<(int)hostInputImageDataGray[(row * imageWidth + col)]<<", ";
        }
        std::cout<<endl;
    }
    */

    // 2. calculate hist 
    dim3 DimGrid2(ceil(imageHeight * imageWidth, BLOCK_WIDTH * BLOCK_WIDTH), 1, 1);
    dim3 DimBlock2(BLOCK_WIDTH * BLOCK_WIDTH, 1, 1);
    hist<<<DimGrid2, DimBlock2>>>(deviceInputImageDataGray, imageWidth * imageHeight, deviceHist);

    // this is for debugging
    /*
    hipDeviceSynchronize();
    unsigned int* hostHist = (unsigned int *) malloc(sizeof(unsigned int) * HISTOGRAM_LENGTH);
    hipMemcpy(hostHist, deviceHist, sizeof(unsigned int) * HISTOGRAM_LENGTH, hipMemcpyDeviceToHost);
    std::cout<<"check image hist "<<std::endl;
    for(int index = 0; index < HISTOGRAM_LENGTH; index++){
        printf("%d, ", hostHist[index]);
    }
    */

    // 3. calculate cdf
    dim3 DimGrid4(1, 1, 1);
    dim3 DimBlock4(HISTOGRAM_LENGTH, 1, 1);
    cal_cdf<<<DimGrid4, DimBlock4>>>(deviceHist, deviceCDF);

    //TODO This is for debugging
    /*
    hipDeviceSynchronize();
    unsigned int * hostCDF = (unsigned int *) malloc(sizeof(unsigned int) * HISTOGRAM_LENGTH);
    hipMemcpy(hostCDF, deviceCDF, sizeof(unsigned int) * HISTOGRAM_LENGTH, hipMemcpyDeviceToHost);
    std::cout<<"check hist CDF "<<std::endl;
    for(int index = 0; index < HISTOGRAM_LENGTH; index++){
        printf("%d, ", hostCDF[index]);
    }
    */
    // 4. histogram equalization

    dim3 DimGrid3(ceil(imageWidth, BLOCK_WIDTH), ceil(imageHeight, BLOCK_WIDTH), 3);
    dim3 DimBlock3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    histogram_equalization<<<DimGrid3, DimBlock3>>>(deviceInputImageData, deviceOutputImageData, deviceCDF, imageHeight, imageWidth);

    hipDeviceSynchronize();
    // 5. memcpy output to host
    hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels, hipMemcpyDeviceToHost);

    // 6. validate the solution
    wbSolution(args, outputImage);

    //@@ insert code here

    // 7. free GPU memory
    hipFree(deviceCDF);
    hipFree(deviceHist);
    hipFree(deviceInputImageData);
    hipFree(deviceInputImageDataGray);
    hipFree(deviceOutputImageData);
   
    // 9. delete image, free cpu memory
    wbImage_delete(inputImage);
    wbImage_delete(outputImage);
    
    return 0;
}


// 0.537255, 0.698039, 0.807843