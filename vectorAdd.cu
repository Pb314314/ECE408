
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for Vector Addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    printf("Hello World from GPU!\n");
}

int main() {
    int n = 1024; // Size of the vectors
    int *a, *b, *c; // Host copies of a, b, c
    int *d_a, *d_b, *d_c; // Device copies of a, b, c
    int size = n * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); 
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    for(int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i*i;
    }

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch vectorAdd() kernel on GPU with N blocks
    vectorAdd<<<(n + 255)/256, 256>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}
