#include "hip/hip_runtime.h"
#include	<wb.h>
#include    <iostream>

#define SEGMENT_LENGTH 256
#define BLOCK_SIZE 256

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < len){
        out[idx] = in1[idx] + in2[idx];
    }
}

int myMin(int a, int b){
    if(a < b){
        return a;
    }
    return b;
}
int ceil(int a, int b){
    return (a + b - 1) / b;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    float* pinnedHostInput1;
    float* pinnedHostInput2;
    float* pinnedHostOutput;

    hipStream_t stream0, stream1, stream2, stream3;
    hipStreamCreate( &stream0);
    hipStreamCreate( &stream1);
    hipStreamCreate( &stream2);
    hipStreamCreate( &stream3);

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    // allocate page-locked memory on CPU 
    hipHostMalloc((void **)&pinnedHostInput1, sizeof(float) * inputLength);
    hipHostMalloc((void **)&pinnedHostInput2, sizeof(float) * inputLength);
    hipHostMalloc((void **)&pinnedHostOutput, sizeof(float) * inputLength);

    // memcpy input
    memcpy(pinnedHostInput1, hostInput1, sizeof(float) * inputLength);
    memcpy(pinnedHostInput2, hostInput2, sizeof(float) * inputLength);



    // 1. Allocate memory on GPU
    hipMalloc((void**) &deviceInput1, sizeof(float) * 4 * SEGMENT_LENGTH);
    hipMalloc((void**) &deviceInput2, sizeof(float) * 4 * SEGMENT_LENGTH);
    hipMalloc((void**) &deviceOutput, sizeof(float) * 4 * SEGMENT_LENGTH);


    dim3 DimGrid(ceil(SEGMENT_LENGTH, BLOCK_SIZE), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // 2. do computation, Breadth First Kernel Issue
    for(int index = 0; index < inputLength; index += 4 * SEGMENT_LENGTH){
        int currentPtr1 = index;
        int currentPtr2 = currentPtr1 + SEGMENT_LENGTH;
        int currentPtr3 = currentPtr2 + SEGMENT_LENGTH;
        int currentPtr4 = currentPtr3 + SEGMENT_LENGTH;
        int length1 = 0, length2 = 0, length3 = 0, length4 = 0;
        
        // copy data
        if(currentPtr1 < inputLength){
            length1 = myMin(SEGMENT_LENGTH, inputLength - currentPtr1);
            hipMemcpyAsync(&deviceInput1[0], &pinnedHostInput1[currentPtr1], sizeof(float) * length1, hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync(&deviceInput2[0], &pinnedHostInput2[currentPtr1], sizeof(float) * length1, hipMemcpyHostToDevice, stream0);
        }
        if(currentPtr2 < inputLength){
            length2 = myMin(SEGMENT_LENGTH, inputLength - currentPtr2);
            hipMemcpyAsync(&deviceInput1[SEGMENT_LENGTH], &pinnedHostInput1[currentPtr2], sizeof(float) * length2, hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(&deviceInput2[SEGMENT_LENGTH], &pinnedHostInput2[currentPtr2], sizeof(float) * length2, hipMemcpyHostToDevice, stream1);
        }
        if(currentPtr3 < inputLength){
            length3 = myMin(SEGMENT_LENGTH, inputLength - currentPtr3);
            hipMemcpyAsync(&deviceInput1[SEGMENT_LENGTH * 2], &pinnedHostInput1[currentPtr3], sizeof(float) * length3, hipMemcpyHostToDevice, stream2);
            hipMemcpyAsync(&deviceInput2[SEGMENT_LENGTH * 2], &pinnedHostInput2[currentPtr3], sizeof(float) * length3, hipMemcpyHostToDevice, stream2);
        }
        if(currentPtr4 < inputLength){
            length4 = myMin(SEGMENT_LENGTH, inputLength - currentPtr4);
            hipMemcpyAsync(&deviceInput1[SEGMENT_LENGTH * 3], &pinnedHostInput1[currentPtr4], sizeof(float) * length4, hipMemcpyHostToDevice, stream3);
            hipMemcpyAsync(&deviceInput2[SEGMENT_LENGTH * 3], &pinnedHostInput2[currentPtr4], sizeof(float) * length4, hipMemcpyHostToDevice, stream3);
        }
        // do calculation
        if(currentPtr1 < inputLength){
            vecAdd<<<DimGrid, DimBlock, 0, stream0>>>(&deviceInput1[0], &deviceInput2[0], &deviceOutput[0], length1);
        }
        if(currentPtr2 < inputLength){
            vecAdd<<<DimGrid, DimBlock, 0, stream1>>>(&deviceInput1[SEGMENT_LENGTH], &deviceInput2[SEGMENT_LENGTH], &deviceOutput[SEGMENT_LENGTH], length2);
        }
        if(currentPtr3 < inputLength){
            vecAdd<<<DimGrid, DimBlock, 0, stream2>>>(&deviceInput1[SEGMENT_LENGTH * 2], &deviceInput2[SEGMENT_LENGTH * 2], &deviceOutput[SEGMENT_LENGTH * 2], length3);
        }
        if(currentPtr4 < inputLength){
            vecAdd<<<DimGrid, DimBlock, 0, stream3>>>(&deviceInput1[SEGMENT_LENGTH * 3], &deviceInput2[SEGMENT_LENGTH * 3], &deviceOutput[SEGMENT_LENGTH * 3], length4);
        }


        // do memory copy from device to host
        if(currentPtr1 < inputLength){
            hipMemcpyAsync(&pinnedHostOutput[currentPtr1], &deviceOutput[0], sizeof(float) * length1, hipMemcpyDeviceToHost, stream0);
        }
        if(currentPtr2 < inputLength){
            hipMemcpyAsync(&pinnedHostOutput[currentPtr2], &deviceOutput[SEGMENT_LENGTH], sizeof(float) * length2, hipMemcpyDeviceToHost, stream1);
        }
        if(currentPtr3 < inputLength){
            hipMemcpyAsync(&pinnedHostOutput[currentPtr3], &deviceOutput[SEGMENT_LENGTH * 2], sizeof(float) * length3, hipMemcpyDeviceToHost, stream2);
        }
        if(currentPtr4 < inputLength){
            hipMemcpyAsync(&pinnedHostOutput[currentPtr4], &deviceOutput[SEGMENT_LENGTH * 3], sizeof(float) * length4, hipMemcpyDeviceToHost, stream3);
        }        
    }
    hipDeviceSynchronize();
    std::cout<<"check hostoutput"<<std::endl;
    for(int index = 0; index < myMin(10, inputLength); index++){
        std::cout<<pinnedHostOutput[index]<<", ";
    }
    std::cout<<std::endl;

    wbSolution(args, pinnedHostOutput, inputLength);

    // free GPU memory
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    // free page-locked memory
    hipHostFree(pinnedHostInput1);
    hipHostFree(pinnedHostInput2);
    hipHostFree(pinnedHostOutput);

    // free pageable memory
    free(hostInput1);
    free(hostInput2);
    

    return 0;
}

