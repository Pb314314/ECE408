#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);      // return the wbarg_t, which contain the input file, output file, expected file and type

    wbTime_start(Generic, "Importing data and creating memory on host");
    // wbArg_getInputFile(args, 0) return the pointer to the first file path
    // wbImport(file path, &inputLength), input file path, return float array, set input length
    // need two input files for this mp. set inputlength using 
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);     // first input float array
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);     // second input float array
    hostOutput = (float *) malloc(inputLength * sizeof(float));                     // output array of float
    wbTime_stop(Generic, "Importing data and creating memory on host");

    // wbLog(kind, message, number);information in log section
    wbLog(TRACE, "The input length is ", inputLength);      

    // wbTime_start(kind, message);information in timer section
	wbTime_start(GPU, "Allocating GPU memory.");
    // section for allocating GPU memory

    // hipMalloc(void** devPtr, size_t size); 
    // first: a pointer to allocated device memory, second: number of bytes to allocate
    // first pointer need to use reference, hipMalloc will modify the pointer to point to the allocated memory on the GPU
    // also need to cast pointer to void** to take the address of any type pointer
    // sizeof return the number of byte per float(return type size_t)
    hipMalloc((void**)deviceInput1, sizeof(float) * inputLength);
    hipMalloc((void**)deviceInput2, sizeof(float) * inputLength);
    hipMalloc((void**)deviceOutput, sizeof(float) * inputLength);
    wbTime_stop(GPU, "Allocating GPU memory.");


    wbTime_start(GPU, "Copying input memory to the GPU.");
    // section for copy input memory from CPU to GPU
    // hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind);
    // don't need explicit cast float* to void*, can implicit cast
    hipMemcpy(deviceInput1, hostInput1, sizeof(float) * inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, sizeof(float) * inputLength, hipMemcpyHostToDevice);
    
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here


    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);
    // free float array
    free(hostInput1);       
    free(hostInput2);       
    free(hostOutput);

    return 0;
}

