#include "hip/hip_runtime.h"
#include    <wb.h>
#include <cmath>

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    // first: calculate the row and col of current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // check thread
    if(row>=0 && row<numCRows && col>=0 && col < numCColumns){
        // perform vector add
        float result = 0;
        for(int i=0;i<numAColumns;i++){
            float A_ele = A[row * numAColumns + i]; // row , i
            float B_ele = B[i * numBColumns + col];                // i, col
            result += A_ele * B_ele;
        }
        C[row*numCColumns + col] = result;
    }
}

#define BLOCK_SIZE 8
/*
int ceil(int a, int b){
    return (int)((a+b-1)/b);
}
*/

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);

    // check whether A's column == B's row
    if(numAColumns != numBRows){
        printf("Dimension Error! Can't do matrix multiplication!\n");
    }
    else printf("Pass the dimension checking. \n");
    //@@ Set numCRows and numCColumns
    numCRows = 0;
    numCColumns = 0;
    numCRows = numARows;
    numCColumns = numBColumns;

    //@@ Allocate the hostC matrix
    hostC = (float *)malloc(sizeof(float) * numCRows * numCColumns);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    // in hipMalloc, give void** device pointer and size of space to malloc in device.
    hipMalloc((void**) &deviceA, sizeof(float) * numARows * numAColumns);
    hipMalloc((void**) &deviceB, sizeof(float) * numBRows * numBColumns);
    hipMalloc((void**) &deviceC, sizeof(float) * numCRows * numCColumns);
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil((float)numCColumns / BLOCK_SIZE), ceil((float)numCRows / BLOCK_SIZE), 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiply<<<DimGrid,DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

